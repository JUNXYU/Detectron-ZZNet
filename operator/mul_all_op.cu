#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "mul_all_op.h"
#include "../core/context_gpu.h"
#include "../core/operator.h"
#include "../core/registry.h"
#include "../core/common.h"
#include "../utils/math.h"


namespace caffe2{

namespace {

    __global__  void Mul_Kernel(const int N, const float* a, const float* b, float* y){
        CUDA_1D_KERNEL_LOOP(index,N){
          y[index] = a[index] * b[index];
        }
    }
}

template<>
bool MulAllOp<float, HIPContext>::RunOnDevice(){

    auto& X = Input(0);
    auto* Y = Output(0);

    vector<TIndex> out_shape;

    for (int i = 0; i < X.ndim() ; ++i) {
    out_shape.push_back(X.dim32(i));
    }
    Y->Resize(out_shape);

    float *output_data = Y->mutable_data<float>();
    const int count = Y->size();
    long nthreads = 256;
    long no_elements = Y->size();
    long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
    long n_yblocks = (long)ceil(
    (float)no_elements / (float)(n_xblocks * nthreads));
    CAFFE_ENFORCE(n_yblocks <= 65535);
    dim3 blocks(n_xblocks, n_yblocks);
    dim3 threads(nthreads);

    for (int i = 0; i < InputSize() ; ++i) {
        if (Y->dims() != Input(i).dims()) {
            CAFFE_THROW(
            "Check failed: output->dims() == Input(i).dims().",
            "Description: Input #",
            i,
            ", input dimension:",
            Input(i).dims(),
            " should match output dimension: ",
            Y->dims()); 
        }
    }
    Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(count, X.data<float>(),Input(1).data<float>(),output_data);
    for (int i = 2; i <InputSize() ; ++i) {
        Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(count, output_data,Input(i).data<float>(),output_data); 
    }
    return true;
}

template <>
    bool MulAllGradientOp<float, HIPContext>::RunOnDevice(){

    auto & X = Input(1);
    auto & dY = Input(0);
    const float *gradOutput_data = dY.data<float>();

    for (int i = 1; i < InputSize() ; ++i) {
        auto& X = Input(i);
        auto* dX = Output(i-1);
        dX->ResizeLike(X);
        float * gradInput_data = dX->mutable_data<float>();

        long no_elements = dX->size();
        // cuda blocks & threads:
        long nthreads = 256;
        // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
        // 65535 for SM 2.x, 2^32 -1 for >= 3.0
        // TODO: When we move to SM 3.5 we should update this
        long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
        long n_yblocks = (long)ceil(
        (float)no_elements / (float)(n_xblocks * nthreads));
        CAFFE_ENFORCE(n_yblocks <= 65535);
        dim3 blocks(n_xblocks, n_yblocks);
        dim3 threads(nthreads);

        if(i == 1){
            math::CopyVector<float, HIPContext >(dX->size(),Input(2).data<float>(),gradInput_data,&context_);
            for (int j = 3; j <InputSize() ; ++j)
            {
                Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(dX->size(),Input(j).data<float>(),gradInput_data,gradInput_data);
            }
              Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(dX->size(),gradInput_data,gradOutput_data,gradInput_data);
        }
         else{
                math::CopyVector<float, HIPContext >(dX->size(),Input(1).data<float>(),gradInput_data,&context_);
                for (int j = 2; j <InputSize() ; ++j) {
                if (i==j){ continue;}
                        Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(dX->size(),Input(j).data<float>(),gradInput_data,gradInput_data);
                }
                Mul_Kernel<<<blocks,threads,0,context_.hip_stream()>>>(dX->size(),gradInput_data,gradOutput_data,gradInput_data);
            }

     }
     return true;

}
REGISTER_HIP_OPERATOR(MulAll,MulAllOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MulAllGradient,MulAllGradientOp<float, HIPContext>);
}